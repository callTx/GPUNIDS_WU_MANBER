#include "hip/hip_runtime.h"
#include"WMGPUinspection.h"

void WMGPUInspection::exec(std::shared_ptr<Packet> pkt)
{



}


WMGPUInspection::WMGPUInspection()
{

}

WMGPUInspection::~WMGPUInspection()
{
}

void WMGPUInspection::cuda_sample_random_ints(int* x,int N){

	for(int i=0; i<N;i++){
		x[i]=rand();
	}
}

void WMGPUInspection::cuda_sample_print_ints(int* x,int N){

	for(int i=0; i<N;i++){
		std::cout<<"x["<<i<<"]: "<<x[i]<<std::endl;
	}
}

__global__ void add(int *a,int *b,int *c,int n){
	//*c= *a+ *b; //Heterogeneous computing
	//c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x]; //Blocks
	//c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x]; //Threads
	
	int index = threadIdx.x + blockIdx.x * blockDim.x; //
	if(index < n) c[index] = a[index] + b[index];
}

int WMGPUInspection::buildMatchingMachine(std::string rules, int bufferSize, int nThreads)
{
		std::cout<<"nThreads: "<<nThreads<<std::endl;
		
		// (14-15/05) getilne -- TmhMenorPadrao() -- padroes.size() -- TmhTodosPadroes() -- BlcStr -- B2=B --  tabeleIdxHash() -- vecPermu() -- TabelaHash() 

		//TODO (20/05) buildMatchingMachine

		std::string line;
    	ifstream file(rules.c_str(), std::ifstream::in);
    	maxs_ = 0;
    	while(getline(file,line))
    	{
        	padroes.push_back(line);    
        	maxs_ += line.size();
		}
		
		m = TmhMenorPadrao();
		nP = padroes.size();
		M = TmhTodosPadroes();
		B = BlcStr(); 
		B2 = B;
		#ifdef VERBOSE
			cout<< "Inicializando tabelas........................... " <<endl;	
		#endif

		tabelaIdxHash();
		tabelaShift();

		return 1;

		/*
		int *a,*b,*c;
		//int a,b,c;
		int *d_a,*d_b,*d_c; 
		// size= sizeof(int);
		//std::cout<<"N: "<<N<<std::endl;
		int size = N * sizeof(int);

		//Aloca espaco para o device copiar de a,b,c
		hipMalloc( (void **)&d_a,size );		
		hipMalloc( (void **)&d_b,size );		
		hipMalloc( (void **)&d_c,size );
		
		a = (int*)malloc(size);
		cuda_sample_random_ints(a,N);
		b= (int*)malloc(size);
		cuda_sample_random_ints(b,N);
		c= (int*)malloc(size);


		//a =2;
		//b =7;
		//copia a,b para o device
		hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);		
		hipMemcpy(d_b,b,size,hipMemcpyHostToDevice);	

		//Launch o kernel add() na GPU
		//add<<<N,1>>>(d_a,d_b,d_c); //blocks
		//add<<<1,N>>>(d_a,d_b,d_c); //threads

		
		//	@ N/THREADS_PER_BLOCK  = blocks number
		
		//add<<<N/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(d_a,d_b,d_c); //threads_per_block with vector multiple of blockDim.x = M 

		add<<<(N+M-1)/M, M >>>(d_a,d_b,d_c,N); //threads_per_block with vector which is not multiple of the blockDim.x = M	

		//Copiar os resultados de volta p/ host
		hipMemcpy(c,d_c,size,hipMemcpyDeviceToHost);

		//cuda_sample_print_ints(c,N);
		//std::cout<<"c: "<<c<<std::endl;
		
		//Cleanup
		hipFree(d_a);
		hipFree(d_b);
		hipFree(d_c);

		//char *hipGetErrorString(hipError_t);
		//printf("%s\n",hipGetErrorString(hipGetLastError()));

		return 0;
		*/
}

int WMGPUInspection::TmhMenorPadrao(){
	
    std::vector<int> sizePadroes;
    for (int i = 0; i <padroes.size() ; ++i) {
        sizePadroes.push_back(padroes[i].size());
    }
    auto minElem = std::min_element(std::begin(sizePadroes), std::end(sizePadroes));
    //std::cout<<minElem.operator*()+"\n"<<std::endl;

   
    return minElem.operator*();
}

int WMGPUInspection::TmhTodosPadroes(){

    int TodosP = m * nP;

    return  TodosP;
}

double WMGPUInspection::BlcStr(){

	double blcStr = log10(2*M)/log10(c);
	//double B = std::ceil(blcStr);
	double Blc = std::round(blcStr);
return Blc;
}

void WMGPUInspection::tabelaIdxHash(){

	vecPermu();
	
	TabelaHash();


}

void WMGPUInspection::vecPermu(){

    vecpermu = std::vector<std::vector<std::string>>(nP,std::vector<std::string>());
 
    for (int i = 0; i < padroes.size() ; ++i) {
        std::string strTemp = padroes[i];
        std::string subStrPerm;

        int idxInicio = 0;
        int idxFim = B -1;
        while(idxFim<strTemp.length()){
            

            subStrPerm = strTemp.substr(idxInicio,B);

            


            if (idxInicio==idxFim){
                
               vecpermu[i].push_back(subStrPerm);
                idxFim = idxFim + 1;
                idxInicio = idxFim;
            }else{
                
                vecpermu[i].push_back(subStrPerm);
               
                idxInicio = idxInicio + 1;
                idxFim = idxFim + 1;

            }

        }

    }


}

void WMGPUInspection::TabelaHash(){
	
	int vpSize = 0;
	for(int j= 0; j< vecpermu.size();j++){
		vpSize = vpSize + vecpermu[j].size();
 	}

 	tabelaidx = std::vector<std::vector<std::string>> (vpSize, std::vector<std::string>());

 	for (int k = 0; k < vecpermu.size(); ++k) {
        	int len  = vecpermu[k].size();
        	for (int i = 0; i < len ; ++i) {
            		int h  = Hash( vecpermu[k].at(i) ,c,M);
            		std::string str = vecpermu[k].at(i);
            		if(h+1<=tabelaidx.size()) {
                		if (std::find(tabelaidx[h].begin(),tabelaidx[h].end(),str) == tabelaidx[h].end() ){
                    			tabelaidx[h].push_back(str);
                		}

            		}else{
                		tabelaidx.resize(h + 1);

                		tabelaidx[h].push_back(str);

            		}

        	}
		
	}

	int sZ = tabelaidx.size()-1;
   	if (tabelaidx[sZ].empty()){
        	tabelaidx[sZ].push_back("*");
	}else{
		tabelaidx.resize(sZ+2);
		tabelaidx[sZ+1].push_back("*");
	}

}

int WMGPUInspection::Hash(std::string strBloco, int c, int M){
    
    int hash = 0;
    for (int i = 0; i <strBloco.length() ; ++i) {
        hash = hash + strBloco[i] * pow(c,i);
	hash = hash % M;
    }

    return hash;
}

void WMGPUInspection::tabelaShift(){
	
    tbShift =  std::vector<int>(tabelaidx.size(),int());
    
    for (int b1 = 0; b1 <tabelaidx.size() ; ++b1) {
        if (!tabelaidx[b1].empty()) {
            for (int b2 = 0; b2 < tabelaidx[b1].size(); ++b2) {
                std::string blocoStr = tabelaidx[b1].at(b2);
                std::vector<int> min;
               
                for (int p = 0; p < padroes.size(); ++p) {
                    

                    std::string pdr;
                    std::vector<char> writable(padroes.at(p).begin(), padroes.at(p).end());
                    writable.push_back('\0');
                    pdr = &*writable.begin();
                    std::string s;

                    int q=0;

                    
                    int idxInicio = 0;
                    int idxFim = B - 1;
                    while (idxFim < pdr.length()){
                        
                        s = pdr.substr(idxInicio,B);
                        if (!blocoStr.find(s)){
                            q = idxFim + 1;

                        }
                        if (idxInicio == idxFim){
                            idxFim = idxFim + 1;
                            idxInicio = idxFim;
                        }else{
                            idxInicio = idxInicio+1;
                            idxFim = idxFim + 1;
                        }
                        if (idxFim == pdr.length() && q > 0){
                            while ((m - q) < 0) {
                                q = q - 1;
                            }

                            min.push_back(m - q);
                        }
                    }
                    

                } 
                if (min.empty()){
                    tbShift.insert(tbShift.begin()+b1,m - B +1);
                    tbShift.erase(tbShift.begin()+b1+1);
		    /*
		    if(b1 == 10542){
		    	#ifdef VERBOSE
				//cout<< ">>>>>>>>>>>tbShift[10542]: "<<tbShift[10542] <<endl;	
	            	#endif
		    }
		    */
                }else {
                    std::vector<int>::iterator result = std::min_element(std::begin(min), std::end(min));
                    tbShift.insert(tbShift.begin() + b1, result.operator*());
                    tbShift.erase(tbShift.begin() + b1 + 1);
		   /* 
		    if(b1 == 10542){
		    	#ifdef VERBOSE
				//cout<< ">>>>>>>>>>>tbShift[10542]: "<<tbShift[10542] <<endl;	
	            	#endif
		    }
		  */
                }
                
            }
        }


    }


}